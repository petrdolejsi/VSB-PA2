#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// OpenGL Graphics includes
#include <glew.h>
#include <freeglut.h>
#include <cudaDefs.h>
#include <imageManager.h>

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_gl.h>      // helper functions for CUDA/GL interop

#define BLOCK_DIM 8

#define COLORS 256

//hipError_t error = hipSuccess;
//hipDeviceProp_t deviceProp = hipDeviceProp_t();

//CUDA variables
unsigned int image_width;
unsigned int image_height;
hipGraphicsResource_t cuda_pbo_resource;
hipGraphicsResource_t cuda_tex_resource;
texture<uchar4, 2, hipReadModeElementType> cuda_tex_ref;
hipChannelFormatDesc cuda_tex_channel_desc;
KernelSetting ks;
unsigned char value = 0;

//backup

uchar4 *d_backup;
size_t backup_pitch;

//OpenGL
unsigned int pbo_id;
unsigned int texture_id;

unsigned int viewport_width = 1024;
unsigned int viewport_height = 1024;

const int size = COLORS * sizeof(float);

float *d_red;
float *d_green;
float *d_blue;

float *d_max;
float *d_searching;

float *h_red;
float *h_green;
float *h_blue;

float h_max[4];
float h_searching[6];

int h_mouse_click[2];
int *d_mouse_click;

int h_viewport_size[2];
int *d_viewport_size;

int h_image_size[2];
int *d_image_size;

bool h_is_selected = false;

#pragma region CUDA Routines

__global__ void apply_filter(int value, int *d_image_size, float *d_red, float *d_green, float *d_blue, int *d_mouse_click, float *d_searching, unsigned char *pbo)
{

	const auto col = (threadIdx.x + blockIdx.x * blockDim.x);
	const auto row = (threadIdx.y + blockIdx.y * blockDim.y);

	const auto offset = col + row * d_image_size[0];
	uchar4 texel = tex2D(cuda_tex_ref, col, row);

	if (d_mouse_click[0] != -1)
	{
		if (texel.y == 255 && texel.z == 255)
		{
			texel.x = value;
		}
	}

	const auto uchar4_pbo = reinterpret_cast<uchar4*>(pbo);

	uchar4_pbo[offset] = texel;
}

__global__ void apply_filter_first_run(int *d_image_size, float *d_red, float *d_green, float *d_blue, uchar4* backup, unsigned int pitch, unsigned char *pbo)
{

	const auto col = (threadIdx.x + blockIdx.x * blockDim.x);
	const auto row = (threadIdx.y + blockIdx.y * blockDim.y);

	const auto offset = col + row * d_image_size[0];
	const uchar4 texel = tex2D(cuda_tex_ref, col, row);

	d_red[texel.x]++;
	d_green[texel.y]++;
	d_blue[texel.z]++;

	const auto offset_backup = col + row * (pitch / 4);
	backup[offset_backup] = texel;

	const auto uchar4_pbo = reinterpret_cast<uchar4*>(pbo);
	uchar4_pbo[offset] = texel;
}

__global__ void apply_filter_restore(int *d_image_size, uchar4* backup, const unsigned int pitch, unsigned char *pbo)
{
	const auto col = (threadIdx.x + blockIdx.x * blockDim.x);
	const auto row = (threadIdx.y + blockIdx.y * blockDim.y);

	const auto offset = col + row * d_image_size[0];
	const auto offset_backup = col + row * (pitch / 4);

	const auto uchar4_pbo = reinterpret_cast<uchar4*>(pbo);
	uchar4_pbo[offset] = backup[offset_backup];
}

__global__ void apply_filter_click(int *d_image_size, float *d_searching, uchar4* backup, const unsigned int pitch, unsigned char *pbo)
{
	const auto col = (threadIdx.x + blockIdx.x * blockDim.x);
	const auto row = (threadIdx.y + blockIdx.y * blockDim.y);

	const auto offset = col + row * d_image_size[0];
	const auto offset_backup = col + row * (pitch / 4);
	auto texel = backup[offset_backup];

	if (texel.x >= d_searching[0] && texel.y >= d_searching[1] && texel.z >= d_searching[2] && texel.x <= d_searching[3] && texel.y <= d_searching[4] && texel.z <= d_searching[5])
	{
		texel.x = 255;
		texel.y = 255;
		texel.z = 255;
	}
	else
	{
		texel.x = texel.x >> 3;
		texel.y = texel.y >> 3;
		texel.z = texel.z >> 3;
	}

	const auto uchar4_pbo = reinterpret_cast<uchar4*>(pbo);

	uchar4_pbo[offset] = texel;
}

__device__ bool check_searched(int value, uchar4 &to_test, uchar4 &result)
{
	if (to_test.y == 255)
	{
		result.x = value;
		result.y = 255;
		result.z = 255;

		return false;
	}
	return true;
}

__global__ void search_neighbourhood (int value, int *d_image_size, unsigned char *pbo)
{

	auto col = (threadIdx.x + blockIdx.x * blockDim.x);
	auto row = (threadIdx.y + blockIdx.y * blockDim.y);

	const auto offset = col + row * d_image_size[0];
	uchar4 texel = tex2D(cuda_tex_ref, col, row);
	uchar4 s = tex2D(cuda_tex_ref, col, row + 1);
	uchar4 es = tex2D(cuda_tex_ref, col + 1, row + 1);
	uchar4 e = tex2D(cuda_tex_ref, col + 1, row);
	uchar4 en = tex2D(cuda_tex_ref, col + 1, row - 1);
	uchar4 n = tex2D(cuda_tex_ref, col, row - 1);
	uchar4 nw = tex2D(cuda_tex_ref, col - 1, row + 1);
	uchar4 w = tex2D(cuda_tex_ref, col - 1, row);
	uchar4 sw = tex2D(cuda_tex_ref, col - 1, row - 1);

	if (check_searched(value, s, texel))
	{
		if (check_searched(value, es, texel))
		{
			if (check_searched(value, e, texel))
			{
				if (check_searched(value, en, texel))
				{
					if (check_searched(value, n, texel))
					{
						if (check_searched(value, nw, texel))
						{
							if (check_searched(value, w, texel))
							{
								check_searched(value, sw, texel);
							}
						}
					}
				}
			}
		}
	}
	
	const auto uchar4_pbo = reinterpret_cast<uchar4*>(pbo);

	uchar4_pbo[offset] = texel;
}

__global__ void search_color(float *d_searching, int *d_mouse_click, int *d_viewport_size, int *d_image_size)
{
	printf("Clicked (viewport): %d %d\n", d_mouse_click[0], d_mouse_click[1]);

	auto mouse_x = (d_mouse_click[0] * d_image_size[0]) / d_viewport_size[0];
	auto mouse_y = d_image_size[1] - (d_mouse_click[1] * d_image_size[1]) / d_viewport_size[1];

	printf("Clicked (image): %d %d\n", mouse_x, d_image_size[1] - mouse_y);

	// ReSharper disable once CppLocalVariableMayBeConst
	uchar4 texel = tex2D(cuda_tex_ref, mouse_x, mouse_y);

	d_searching[0] = texel.x;
	d_searching[1] = texel.y;
	d_searching[2] = texel.z;

	d_searching[3] = texel.x;
	d_searching[4] = texel.y;
	d_searching[5] = texel.z;

	printf("Selected color: %d %d %d\n", texel.x, texel.y, texel.z);
}

__global__ void draw_histogram(const unsigned int histogram_height, const unsigned int dst_pitch, float *d_red, float *d_green, float *d_blue, float *d_max, uchar4* dst)
{
	const auto col = (threadIdx.x + blockIdx.x * blockDim.x);
	const auto row = (threadIdx.y + blockIdx.y * blockDim.y);

	const auto offset4 = col + row * (dst_pitch / 4);
	const auto offset3 = col + row * (dst_pitch / 4) + histogram_height * (dst_pitch / 4);
	const auto offset2 = col + row * (dst_pitch / 4) + histogram_height * (dst_pitch / 4) * 2;
	const auto offset1 = col + row * (dst_pitch / 4) + histogram_height * (dst_pitch / 4) * 3;
	uchar4 texel1;

	texel1.w = 255;
	texel1.x = 0;
	texel1.y = 0;
	texel1.z = 0;

	auto texel2 = texel1;
	auto texel3 = texel1;
	auto texel4 = texel1;

	const auto height_red = (histogram_height * d_red[col]) / d_max[0];
	if (height_red > row)
	{
		texel2.z = 255;
	}

	const auto height_green = (histogram_height * d_green[col]) / d_max[1];
	if (height_green > row)
	{
		texel3.y = 255;
	}

	const auto height_blue = (histogram_height * d_blue[col]) / d_max[2];
	if (height_blue > row)
	{
		texel4.x = 255;
	}

	const auto height_red_rgb = (histogram_height * d_red[col]) / d_max[3];
	if (height_red_rgb > row)
	{
		texel1.z = 255;
	}

	const auto height_green_rgb = (histogram_height * d_green[col]) / d_max[3];
	if (height_green_rgb > row)
	{
		texel1.y = 255;
	}

	const auto height_blue_rgb = (histogram_height * d_blue[col]) / d_max[3];
	if (height_blue_rgb > row)
	{
		texel1.x = 255;
	}

	dst[offset1] = texel1;
	dst[offset2] = texel2;
	dst[offset3] = texel3;
	dst[offset4] = texel4;
}

void mouse_click(const int button, const int state, const int x, const int y)
{

	if (!h_is_selected && button == GLUT_LEFT_BUTTON && state == GLUT_DOWN)
	{
		h_mouse_click[0] = x;
		h_mouse_click[1] = y;

		hipArray* array;
		checkCudaErrors(hipGraphicsMapResources(1, &cuda_tex_resource, nullptr));
		checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&array, cuda_tex_resource, 0, 0));
		checkCudaErrors(hipGetChannelDesc(&cuda_tex_channel_desc, array));
		checkCudaErrors(hipBindTextureToArray(&cuda_tex_ref, array, &cuda_tex_channel_desc));

		unsigned char *pbo_data;
		size_t pbo_size;
		checkCudaErrors(hipGraphicsMapResources(1, &cuda_pbo_resource, nullptr));

		checkCudaErrors(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&pbo_data), &pbo_size, cuda_pbo_resource));

		checkCudaErrors(hipMemcpy(d_mouse_click, h_mouse_click, sizeof(int) * 2, hipMemcpyHostToDevice));

		search_color << < 1, 1 >> > (d_searching, d_mouse_click, d_viewport_size, d_image_size);

		apply_filter_click << <ks.dimGrid, ks.dimBlock >> > (d_image_size, d_searching, d_backup, backup_pitch, pbo_data);

		checkCudaErrors(hipUnbindTexture(&cuda_tex_ref));
		checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_pbo_resource, nullptr));
		checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_tex_resource, nullptr));

		glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo_id);
		glBindTexture(GL_TEXTURE_2D, texture_id);
		glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, image_width, image_height, GL_RGBA, GL_UNSIGNED_BYTE, nullptr);

		h_is_selected = true;
	}
}

void save_histogram(const char *filename)
{
	uchar4 *d_result;
	size_t result_pitch;

	const auto histogram_width = 255;
	const auto histogram_height = 200;

	checkCudaErrors(hipMallocPitch(&d_result, &result_pitch, histogram_width * 4, histogram_height * 4));

	KernelSetting histogram_ks;

	histogram_ks.dimBlock = dim3(BLOCK_DIM, BLOCK_DIM, 1);
	histogram_ks.blockSize = BLOCK_DIM * BLOCK_DIM;
	histogram_ks.dimGrid = dim3((histogram_width + BLOCK_DIM - 1) / BLOCK_DIM, (histogram_height + BLOCK_DIM - 1) / BLOCK_DIM, 1);

	checkCudaErrors(hipMemcpy(h_red, d_red, size, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(h_green, d_green, size, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(h_blue, d_blue, size, hipMemcpyDeviceToHost));
		
	for (auto i = 0; i < 256; i++)
	{
		if (h_red[i] > h_max[0])
		{
			h_max[0] = h_red[i];
		}

		if (h_green[i] > h_max[1])
		{
			h_max[1] = h_green[i];
		}

		if (h_blue[i] > h_max[2])
		{
			h_max[2] = h_blue[i];
		}
	}
	h_max[3] = h_max[0];

	if (h_max[1] > h_max[3])
		h_max[3] = h_max[1];

	if (h_max[2] > h_max[3])
		h_max[3] = h_max[2];

	checkCudaErrors(hipMalloc(&d_max, 4 * sizeof(float)));
	checkCudaErrors(hipMemcpy(d_max, h_max, 4 * sizeof(float), hipMemcpyHostToDevice));

	draw_histogram << <histogram_ks.dimGrid, histogram_ks.dimBlock >> > (histogram_height, result_pitch, d_red, d_green, d_blue, d_max, d_result);

	// ReSharper disable once CppUseAuto
	FIBITMAP *h_image = FreeImage_Allocate(histogram_width, histogram_height * 4, 32);

	checkCudaErrors(hipMemcpy2D(FreeImage_GetBits(h_image), FreeImage_GetPitch(h_image), d_result, result_pitch, histogram_width * 4, histogram_height * 4, hipMemcpyDeviceToHost));

	FreeImage_Save(FIF_PNG, h_image, filename, 0);

	FreeImage_Unload(h_image);

	checkCudaErrors(hipFree(d_result));

	printf("Saved histogram as %s\n", filename);
}

void make_founds_bigger()
{
	if (!h_is_selected)
	{
		return;
	}
	
	hipArray* array;
	checkCudaErrors(hipGraphicsMapResources(1, &cuda_tex_resource, nullptr));
	checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&array, cuda_tex_resource, 0, 0));
	checkCudaErrors(hipGetChannelDesc(&cuda_tex_channel_desc, array));
	checkCudaErrors(hipBindTextureToArray(&cuda_tex_ref, array, &cuda_tex_channel_desc));

	unsigned char *pbo_data;
	size_t pbo_size;
	checkCudaErrors(hipGraphicsMapResources(1, &cuda_pbo_resource, nullptr));

	checkCudaErrors(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&pbo_data), &pbo_size, cuda_pbo_resource));

	search_neighbourhood << <ks.dimGrid, ks.dimBlock >> > (value, d_image_size, pbo_data);

	checkCudaErrors(hipUnbindTexture(&cuda_tex_ref));
	checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_pbo_resource, nullptr));
	checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_tex_resource, nullptr));

	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo_id);
	glBindTexture(GL_TEXTURE_2D, texture_id);
	glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, image_width, image_height, GL_RGBA, GL_UNSIGNED_BYTE, nullptr);
}

void restore_image()
{
	hipArray* array;

	checkCudaErrors(hipGraphicsMapResources(1, &cuda_tex_resource, nullptr));
	checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&array, cuda_tex_resource, 0, 0));
	checkCudaErrors(hipGetChannelDesc(&cuda_tex_channel_desc, array));
	checkCudaErrors(hipBindTextureToArray(&cuda_tex_ref, array, &cuda_tex_channel_desc));

	unsigned char *pbo_data;
	size_t pbo_size;
	checkCudaErrors(hipGraphicsMapResources(1, &cuda_pbo_resource, nullptr));

	checkCudaErrors(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&pbo_data), &pbo_size, cuda_pbo_resource));

	apply_filter_restore<< <ks.dimGrid, ks.dimBlock >> > (d_image_size, d_backup, backup_pitch, pbo_data);

	checkCudaErrors(hipUnbindTexture(&cuda_tex_ref));
	checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_pbo_resource, nullptr));
	checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_tex_resource, nullptr));

	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo_id);
	glBindTexture(GL_TEXTURE_2D, texture_id);
	glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, image_width, image_height, GL_RGBA, GL_UNSIGNED_BYTE, nullptr);   //Source parameter is NULL, Data is coming from a PBO, not host memory

	h_mouse_click[0] = h_mouse_click[1] = -1;
	checkCudaErrors(hipMemcpy(d_mouse_click, h_mouse_click, sizeof(int) * 2, hipMemcpyHostToDevice));

	h_is_selected = false;

	printf("Restored image\n\n");
}

void change_searching_range()
{
	if (!h_is_selected)
	{
		return;
	}
	
	checkCudaErrors(hipMemcpy(h_searching, d_searching, 6 * sizeof(float), hipMemcpyDeviceToHost));
	if (h_searching[0] >= 1)
	{
		h_searching[0]--;
	}
	if (h_searching[1] >= 1)
	{
		h_searching[1]--;
	}
	if (h_searching[2] >= 1)
	{
		h_searching[2]--;
	}

	if (h_searching[3] <= 254)
	{
		h_searching[3]++;
	}
	if (h_searching[4] <= 254)
	{
		h_searching[4]++;
	}
	if (h_searching[5] <= 254)
	{
		h_searching[5]++;
	}
	checkCudaErrors(hipMemcpy(d_searching, h_searching, 6 * sizeof(float), hipMemcpyHostToDevice));

	hipArray* array;

	checkCudaErrors(hipGraphicsMapResources(1, &cuda_tex_resource, nullptr));
	checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&array, cuda_tex_resource, 0, 0));
	checkCudaErrors(hipGetChannelDesc(&cuda_tex_channel_desc, array));
	checkCudaErrors(hipBindTextureToArray(&cuda_tex_ref, array, &cuda_tex_channel_desc));

	unsigned char *pbo_data;
	size_t pbo_size;
	checkCudaErrors(hipGraphicsMapResources(1, &cuda_pbo_resource, nullptr));

	checkCudaErrors(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&pbo_data), &pbo_size, cuda_pbo_resource));

	apply_filter_click << <ks.dimGrid, ks.dimBlock >> > (d_image_size, d_searching, d_backup, backup_pitch, pbo_data);

	checkCudaErrors(hipUnbindTexture(&cuda_tex_ref));
	checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_pbo_resource, nullptr));
	checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_tex_resource, nullptr));

	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo_id);
	glBindTexture(GL_TEXTURE_2D, texture_id);
	glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, image_width, image_height, GL_RGBA, GL_UNSIGNED_BYTE, nullptr);   //Source parameter is NULL, Data is coming from a PBO, not host memory

	printf("Changed range of searched color\n");
}

void keyboard(const unsigned char key, int x, int y)
{
	if (key == 27)
	{
		exit(EXIT_SUCCESS);
	}

	if (key == 's')
	{
		save_histogram("histogram.png");
	}
	
	if (key == 'm')
	{
		make_founds_bigger();
	}
	
	if (key == 'r')
	{
		restore_image();
	}

	if (key == 'c')
	{
		change_searching_range();
	}
}

void cuda_worker_first_run()
{
	hipArray* array;

	checkCudaErrors(hipGraphicsMapResources(1, &cuda_tex_resource, nullptr));
	checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&array, cuda_tex_resource, 0, 0));
	checkCudaErrors(hipGetChannelDesc(&cuda_tex_channel_desc, array));
	checkCudaErrors(hipBindTextureToArray(&cuda_tex_ref, array, &cuda_tex_channel_desc));

	unsigned char *pbo_data;
	size_t pbo_size;
	checkCudaErrors(hipGraphicsMapResources(1, &cuda_pbo_resource, nullptr));

	checkCudaErrors(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&pbo_data), &pbo_size, cuda_pbo_resource));

	checkCudaErrors(hipMallocPitch(&d_backup, &backup_pitch, image_width * 4, image_height));

	ks.blockSize = BLOCK_DIM * BLOCK_DIM;
	ks.dimBlock = dim3(BLOCK_DIM, BLOCK_DIM, 1);
	ks.dimGrid = dim3((image_width + BLOCK_DIM - 1) / BLOCK_DIM, (image_height + BLOCK_DIM - 1) / BLOCK_DIM, 1);

	apply_filter_first_run << <ks.dimGrid, ks.dimBlock >> > (d_image_size, d_red, d_green, d_blue, d_backup, backup_pitch, pbo_data);

	checkCudaErrors(hipUnbindTexture(&cuda_tex_ref));
	checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_pbo_resource, nullptr));
	checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_tex_resource, nullptr));

	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo_id);
	glBindTexture(GL_TEXTURE_2D, texture_id);
	glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, image_width, image_height, GL_RGBA, GL_UNSIGNED_BYTE, nullptr);   //Source parameter is NULL, Data is coming from a PBO, not host memory

	checkCudaErrors(hipMemcpy(h_red, d_red, size, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(h_green, d_green, size, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(h_blue, d_blue, size, hipMemcpyDeviceToHost));

	h_mouse_click[0] = -1;
	checkCudaErrors(hipMemcpy(d_mouse_click, h_mouse_click, sizeof(int) * 2, hipMemcpyHostToDevice));
}

void cuda_worker()
{
	hipArray* array;

	//T ODO 3: Map cudaTexResource
	checkCudaErrors(hipGraphicsMapResources(1, &cuda_tex_resource, nullptr));

	//T ODO 4: Get Mapped Array of cudaTexResource
	checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&array, cuda_tex_resource, 0, 0));

	//T ODO 5: Get cudaTexChannelDesc from previously obtained array
	checkCudaErrors(hipGetChannelDesc(&cuda_tex_channel_desc, array));

	//T ODO 6: Bind cudaTexRef to array
	checkCudaErrors(hipBindTextureToArray(&cuda_tex_ref, array, &cuda_tex_channel_desc));

	unsigned char *pbo_data;
	size_t pbo_size;
	//T ODO 7: Map cudaPBOResource
	checkCudaErrors(hipGraphicsMapResources(1, &cuda_pbo_resource, nullptr));

	//T ODO 7: Map Mapped pointer to cudaPBOResource data
	checkCudaErrors(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&pbo_data), &pbo_size, cuda_pbo_resource));

	//T ODO 8: Set KernelSetting variable ks (dimBlock, dimGrid, etc.) such that block will have BLOCK_DIM x BLOCK_DIM threads
	//done in cuda_worker_first_run

	//Calling applyFilter kernel
	value-=2;
	if (value < 10) value = 255;

	apply_filter << <ks.dimGrid, ks.dimBlock >> > (value, d_image_size, d_red, d_green, d_blue, d_mouse_click, d_searching, pbo_data);

	//Following code release mapped resources, unbinds texture and ensures that PBO data will be copied into OpenGL texture. Do not modify following code!
	checkCudaErrors(hipUnbindTexture(&cuda_tex_ref));
	checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_pbo_resource, nullptr));
	checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_tex_resource, nullptr));

	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo_id);
	glBindTexture(GL_TEXTURE_2D, texture_id);
	glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, image_width, image_height, GL_RGBA, GL_UNSIGNED_BYTE, nullptr);   //Source parameter is NULL, Data is coming from a PBO, not host memory
}

void init_cuda_tex()
{
	// ReSharper disable once CppDeprecatedEntity
	cudaGLSetGLDevice(0);
	checkError();

	//T ODO 1: Register OpenGL texture to CUDA resource
	checkCudaErrors(hipGraphicsGLRegisterImage(&cuda_tex_resource, texture_id, GL_TEXTURE_2D, hipGraphicsRegisterFlagsReadOnly));

	//CUDA Texture settings
	cuda_tex_ref.normalized = false;						//Otherwise TRUE to access with normalized texture coordinates
	cuda_tex_ref.filterMode = hipFilterModePoint;			//Otherwise texRef.filterMode = hipFilterModeLinear; for Linear interpolation of texels
	cuda_tex_ref.addressMode[0] = hipAddressModeClamp;		//No repeat texture pattern
	cuda_tex_ref.addressMode[1] = hipAddressModeClamp;		//No repeat texture pattern

	//T ODO 2: Register PBO to CUDA resource
	checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_pbo_resource, pbo_id, hipGraphicsRegisterFlagsWriteDiscard));
}

void release_cuda()
{
	checkCudaErrors(hipFree(d_backup));
	
	checkCudaErrors(hipFree(d_red));
	checkCudaErrors(hipFree(d_green));
	checkCudaErrors(hipFree(d_blue));
	checkCudaErrors(hipFree(d_max));
	checkCudaErrors(hipFree(d_searching));
	checkCudaErrors(hipFree(d_mouse_click));
	checkCudaErrors(hipFree(d_viewport_size));
	checkCudaErrors(hipFree(d_image_size));
	
	checkCudaErrors(hipGraphicsUnregisterResource(cuda_pbo_resource));
	checkCudaErrors(hipGraphicsUnregisterResource(cuda_tex_resource));
}
#pragma endregion

#pragma region OpenGL Routines - DO NOT MODIFY THIS SECTION !!!

void load_texture(const char* image_file_name)
{
	FreeImage_Initialise();
	const auto temp = ImageManager::GenericLoader(image_file_name, 0);

	image_width = FreeImage_GetWidth(temp);
	image_height = FreeImage_GetHeight(temp);

	//OpenGL Texture
	glEnable(GL_TEXTURE_2D);
	glGenTextures(1, &texture_id);
	glBindTexture(GL_TEXTURE_2D, texture_id);

	//WARNING: Just some of inner format are supported by CUDA!!!
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, image_width, image_height, 0, GL_BGRA, GL_UNSIGNED_BYTE, FreeImage_GetBits(temp));
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);

	FreeImage_Unload(temp);

	h_red = static_cast<float*>(malloc(size));
	h_green = static_cast<float*>(malloc(size));
	h_blue = static_cast<float*>(malloc(size));

	for (auto i = 0; i < COLORS; i++)
	{
		h_red[i] = 0;
		h_green[i] = 0;
		h_blue[i] = 0;
	}

	checkCudaErrors(hipMalloc(&d_red, size));
	checkCudaErrors(hipMalloc(&d_green, size));
	checkCudaErrors(hipMalloc(&d_blue, size));
	checkCudaErrors(hipMalloc(&d_searching, 6 * sizeof(float)));

	checkCudaErrors(hipMemcpy(d_red, h_red, size, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_green, h_green, size, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_blue, h_blue, size, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_searching, h_searching, 6 * sizeof(float), hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc(&d_mouse_click, sizeof(int) * 2));

	h_mouse_click[0] = h_mouse_click[1] = -1;
	checkCudaErrors(hipMemcpy(d_mouse_click, h_mouse_click, sizeof(int) * 2, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc(&d_viewport_size, sizeof(int) * 2));
	h_viewport_size[0] = viewport_width;
	h_viewport_size[1] = viewport_height;
	checkCudaErrors(hipMemcpy(d_viewport_size, h_viewport_size, sizeof(int) * 2, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc(&d_image_size, sizeof(int) * 2));
	h_image_size[0] = image_width;
	h_image_size[1] = image_height;
	checkCudaErrors(hipMemcpy(d_image_size, h_image_size, sizeof(int) * 2, hipMemcpyHostToDevice));
}

void prepare_pbo()
{
	glGenBuffers(1, &pbo_id);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo_id);												// Make this the current UNPACK buffer (OpenGL is state-based)
	glBufferData(GL_PIXEL_UNPACK_BUFFER, image_width * image_height * 4, nullptr, GL_DYNAMIC_COPY);	// Allocate data for the buffer. 4-channel 8-bit image
}

void my_display()
{
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	glEnable(GL_TEXTURE_2D);
	glBindTexture(GL_TEXTURE_2D, texture_id);

	//I know this is a very old OpenGL, but we want to practice CUDA :-)
		//Now it will be a wasted time to learn you current features of OpenGL. Sorry for that however, you can visit my second seminar dealing with Computer Graphics (CG2).
	glBegin(GL_QUADS);

	glTexCoord2d(0, 0);		glVertex2d(0, 0);
	glTexCoord2d(1, 0);		glVertex2d(viewport_width, 0);
	glTexCoord2d(1, 1);		glVertex2d(viewport_width, viewport_height);
	glTexCoord2d(0, 1);		glVertex2d(0, viewport_height);

	glEnd();

	glDisable(GL_TEXTURE_2D);

	glFlush();
	glutSwapBuffers();
}

void my_resize(const GLsizei w, const GLsizei h)
{
	viewport_width = w;
	viewport_height = h;

	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	glViewport(0, 0, viewport_width, viewport_height);

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluOrtho2D(0, viewport_width, 0, viewport_height);

	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	glutPostRedisplay();

	h_viewport_size[0] = w;
	h_viewport_size[1] = h;
	checkCudaErrors(hipMemcpy(d_viewport_size, h_viewport_size, sizeof(int) * 2, hipMemcpyHostToDevice));
}

void my_idle()
{
	cuda_worker();
	glutPostRedisplay();
}

void init_gl(int argc, char **argv)
{
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
	glutInitWindowSize(viewport_width, viewport_height);
	glutInitWindowPosition(0, 0);
	glutCreateWindow("Image");

	glutDisplayFunc(my_display);
	glutReshapeFunc(my_resize);
	glutIdleFunc(my_idle);
	glutMouseFunc(mouse_click);
	glutKeyboardFunc(keyboard);
	glutSetCursor(GLUT_CURSOR_CROSSHAIR);

	// initialize necessary OpenGL extensions
	glewInit();

	glClearColor(0.0, 0.0, 0.0, 1.0);
	glShadeModel(GL_SMOOTH);
	glViewport(0, 0, viewport_width, viewport_height);

	glFlush();
}

void release_open_gl()
{
	if (texture_id > 0)
		glDeleteTextures(1, &texture_id);
	if (pbo_id > 0)
		glDeleteBuffers(1, &pbo_id);
}

#pragma endregion

void release_resources()
{
	release_cuda();
	release_open_gl();
}

void project(const int argc, char *argv[])
{
	//initializeCUDA(deviceProp);

	init_gl(argc, argv);

	load_texture("lena.png");
	//load_texture("image.tif");
	//load_texture("testing.png");

	prepare_pbo();

	init_cuda_tex();

	cuda_worker_first_run();

	printf("------------------------------------------------------------------------\n\n");
	printf("To select color, use the cursor and click somewhere on the image\n");
	printf("Keyboard shortcuts: \n\t s - create and Save histogram (histogram.png) \n\t m - Make founds (if exist) bigger \n\t r - Restore image (hide founds and make image brighter) \n\t c - Change range of searched color by +-1 \n\t ESC - close image and terminate program\n\n");
	printf("------------------------------------------------------------------------\n");

	//start rendering main loop
	glutMainLoop();
	atexit(release_resources);
}
