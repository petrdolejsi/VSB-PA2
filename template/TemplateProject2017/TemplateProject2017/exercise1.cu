#include "hip/hip_runtime.h"
#include <cudaDefs.h>
#include <time.h>
#include "exercise1.cuh"

constexpr unsigned int THREADS_PER_BLOCK = 256;
constexpr unsigned int MEMBLOCK_PER_THREADBLOCK = 2;

void exercise1()
{
	part1_1();
	//part1_2();
}

void part1_1()
{
	srand(time(NULL));
	
	printf("Exercise 1 - Part 1\n");
	
	const unsigned int m = 5000;
	const unsigned int size = m * sizeof(int);

	auto a_host = static_cast<int*>(malloc(size));
	auto b_host = static_cast<int*>(malloc(size));
	auto c_host = static_cast<int*>(malloc(size));

	for (auto i = 0; i < m; i++)
	{
		a_host[i] = rand();
		b_host[i] = rand();
	}

	int *a_device;
	int *b_device;
	int *c_device;

	hipEvent_t startEvent, stopEvent;
	float elapsedTime;
	hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);
	hipEventRecord(startEvent, 0);

	checkCudaErrors(hipMalloc(&a_device, size));
	checkCudaErrors(hipMalloc(&b_device, size));
	checkCudaErrors(hipMalloc(&c_device, size));

	checkCudaErrors(hipMemcpy(a_device, a_host, size, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(b_device, b_host, size, hipMemcpyHostToDevice));

	vector_add_m <<< 1, m >>> (a_device, b_device, c_device, m);

	checkCudaErrors(hipMemcpy(c_host, c_device, size, hipMemcpyDeviceToHost));

	checkCudaErrors(hipFree(a_device));
	checkCudaErrors(hipFree(b_device));
	checkCudaErrors(hipFree(c_device));

	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);
	hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
	hipEventDestroy(startEvent);
	hipEventDestroy(stopEvent);

	for (auto i = 0; i < m; i++)
	{
		std::cout << c_host[i] << std::endl;
	}

	printf("Time to get device properties: %f ms", elapsedTime);

	free(a_host);
	free(b_host);
	free(c_host);

	a_host = nullptr;
	b_host = nullptr;
	c_host = nullptr;
}

void part1_2()
{
	printf("Exercise 1 - Part 2\n");
	
	const unsigned int m = 20;
	const unsigned int n = 20;

	auto *a_host = static_cast<int*>(malloc(m * n * sizeof(int)));
	auto *b_host = static_cast<int*>(malloc(m * n * sizeof(int)));
	auto *c_host = static_cast<int*>(malloc(m * n * sizeof(int)));

	for (auto i = 0; i < m; i++)
	{
		for (auto j = 0; j < n; j++)
		{
			a_host[i * m + j] = (i + 1) * (j + 1);
			b_host[i * m + j] = (i + 1) * (j + 1);
		}
	}

	int *a_device;
	int *b_device;
	int *c_device;

	checkCudaErrors(hipMalloc(&a_device, sizeof(int) * m * n));
	checkCudaErrors(hipMalloc(&b_device, sizeof(int) * m * n));
	checkCudaErrors(hipMalloc(&c_device, sizeof(int) * m * n));
	
	checkCudaErrors(hipMemcpy(a_device, a_host, m * n * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(b_device, b_host, m * n * sizeof(int), hipMemcpyHostToDevice));

	dim3 dimBlock(m, n);
	dim3 dimGrid(1, 1);

	vector_add_n_m <<<dimGrid, dimBlock >>> (a_device, b_device, c_device, m, n);

	checkCudaErrors(hipMemcpy(c_host, c_device, m * n * sizeof(int), hipMemcpyDeviceToHost));

	for (int i = 0; i < m; i++)
	{
		std::cout << i << ": " ;
		for (int j = 0; j < n; j++)
		{
			std::cout << c_host[i * m + j] << "  ";
		}
		std::cout << std::endl;
	}

	checkCudaErrors(hipFree(a_device));
	checkCudaErrors(hipFree(b_device));
	checkCudaErrors(hipFree(c_device));

	free(a_host);
	free(b_host);
	free(c_host);

	a_host = nullptr;
	b_host = nullptr;
	c_host = nullptr;
}

__global__ void vector_add_m (int *a, int *b, int *c, const int m)
{
	const int i = threadIdx.x;
	if (i < m)
	{
		c[i] = a[i] + b[i];
	}
}

__global__ void vector_add_n_m (int *a, int *b, int *c, const int m, const int n)
{
	const auto i = blockIdx.x * blockDim.x + threadIdx.x;
	const auto j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < m && j < n)
	{
		c[i * m + j] = a[i * m + j] + b[i * m + j];
	}
}