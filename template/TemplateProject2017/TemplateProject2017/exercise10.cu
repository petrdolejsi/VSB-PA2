#include "hip/hip_runtime.h"
#include <cudaDefs.h>
#include <time.h>
#include <math.h>


const unsigned int N = 1 << 20;
const unsigned int MEMSIZE = N * sizeof(unsigned int);
const unsigned int NO_LOOPS = 100;
const unsigned int THREAD_PER_BLOCK = 256;
const unsigned int GRID_SIZE = (N + THREAD_PER_BLOCK - 1)/THREAD_PER_BLOCK;

void fillData(unsigned int *data, const unsigned int length)
{
	//srand(time(0));
	for (unsigned int i=0; i<length; i++)
	{
		//data[i]= rand();
		data[i]= 1;
	}
}

void printData(const unsigned int *data, const unsigned int length)
{
	if (data ==0) return;
	for (unsigned int i=0; i<length; i++)
	{
		printf("%u ", data[i]);
	}
}


__global__ void kernel(const unsigned int *a, const unsigned int *b, const unsigned int length, unsigned int *c)
{
	const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	//TODO:  thread block loop
	if (tid < length)
	{
		c[tid] = a[tid] + b[tid];
	}
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Tests 1. - single stream, async calling </summary>
////////////////////////////////////////////////////////////////////////////////////////////////////
void test1()
{
	hipStream_t stream;
	hipStreamCreate(&stream);
	unsigned int *a, *b, *c;
	unsigned int *da, *db, *dc;

	// paged-locked allocation
	hipHostAlloc((void**)&a, NO_LOOPS * MEMSIZE,hipHostMallocDefault);
	hipHostAlloc((void**)&b, NO_LOOPS * MEMSIZE,hipHostMallocDefault);
	hipHostAlloc((void**)&c, NO_LOOPS * MEMSIZE,hipHostMallocDefault);

	fillData(a, NO_LOOPS * N);
	fillData(b, NO_LOOPS * N);

	// Data chunks on GPU
	hipMalloc((void**)&da, MEMSIZE);
	hipMalloc((void**)&db, MEMSIZE);
	hipMalloc((void**)&dc, MEMSIZE);

	//T ODO: create stream
	
	unsigned int dataOffset = 0;

	hipEvent_t startEvent, stopEvent;
	float elapsedTime;
	hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);
	hipEventRecord(startEvent, 0);

	for(int i=0; i < NO_LOOPS; i++)
	{
		//TODO:  copy a->da, b->db
		//TODO:  run the kernel in the stream
		//TODO:  copy dc->c
		hipMemcpyAsync(da, &a[dataOffset], MEMSIZE, hipMemcpyHostToDevice, stream);
		hipMemcpyAsync(db, &b[dataOffset], MEMSIZE, hipMemcpyHostToDevice, stream);
		kernel << <GRID_SIZE, THREAD_PER_BLOCK, 0, stream >> > (da, db, N, dc);
		hipMemcpyAsync(&c[dataOffset], dc, MEMSIZE, hipMemcpyDeviceToHost, stream);

		dataOffset += N;
	}

	//TODO: Synchonize stream

	hipStreamSynchronize(stream);

	//TODO: Destroy stream

	hipStreamDestroy(stream);

	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);
	hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
	printf("Test 1 time: %f ms\n", elapsedTime);

	printData(c, 100);
	
	hipFree(da);
	hipFree(db);
	hipFree(dc);

	hipHostFree(a);
	hipHostFree(b);
	hipHostFree(c);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Tests 2. - two streams - depth first approach </summary>
////////////////////////////////////////////////////////////////////////////////////////////////////
void test2()
{
	//TODO: reuse the source code of above mentioned method test1()
	hipStream_t stream0;
	hipStream_t stream1;
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);
	unsigned int *a, *b, *c;
	unsigned int *da, *db, *dc;

	// paged-locked allocation
	hipHostAlloc((void**)&a, NO_LOOPS * MEMSIZE, hipHostMallocDefault);
	hipHostAlloc((void**)&b, NO_LOOPS * MEMSIZE, hipHostMallocDefault);
	hipHostAlloc((void**)&c, NO_LOOPS * MEMSIZE, hipHostMallocDefault);

	fillData(a, NO_LOOPS * N);
	fillData(b, NO_LOOPS * N);

	// Data chunks on GPU
	hipMalloc((void**)&da, MEMSIZE);
	hipMalloc((void**)&db, MEMSIZE);
	hipMalloc((void**)&dc, MEMSIZE);

	//T ODO: create stream

	unsigned int dataOffset0 = 0;
	unsigned int dataOffset1 = N;

	hipEvent_t startEvent, stopEvent;
	float elapsedTime;
	hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);
	hipEventRecord(startEvent, 0);

	for (int i = 0; i < NO_LOOPS; i+=2)
	{
		//TODO:  copy a->da, b->db
		//TODO:  run the kernel in the stream
		//TODO:  copy dc->c
		hipMemcpyAsync(da, &a[dataOffset0], MEMSIZE, hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(db, &b[dataOffset0], MEMSIZE, hipMemcpyHostToDevice, stream0);
		kernel << <GRID_SIZE, THREAD_PER_BLOCK, 0, stream0 >> > (da, db, N, dc);
		hipMemcpyAsync(&c[dataOffset0], dc, MEMSIZE, hipMemcpyDeviceToHost, stream0);

		dataOffset0 += (2 * N);

		hipMemcpyAsync(da, &a[dataOffset1], MEMSIZE, hipMemcpyHostToDevice, stream1);
		hipMemcpyAsync(db, &b[dataOffset1], MEMSIZE, hipMemcpyHostToDevice, stream1);
		kernel << <GRID_SIZE, THREAD_PER_BLOCK, 0, stream1 >> > (da, db, N, dc);
		hipMemcpyAsync(&c[dataOffset1], dc, MEMSIZE, hipMemcpyDeviceToHost, stream1);

		dataOffset1 += (2 * N);
	}

	//TODO: Synchonize stream

	hipStreamSynchronize(stream0);
	hipStreamSynchronize(stream1);

	//TODO: Destroy stream

	hipStreamDestroy(stream0);
	hipStreamDestroy(stream1);

	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);
	hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
	printf("\nTest 2 time: %f ms\n", elapsedTime);

	printData(c, 100);

	hipFree(da);
	hipFree(db);
	hipFree(dc);

	hipHostFree(a);
	hipHostFree(b);
	hipHostFree(c);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Tests 3. - two streams - breadth first approach</summary>
////////////////////////////////////////////////////////////////////////////////////////////////////
void test3()
{
	//TODO: reuse the source code of above mentioned method test1()
	hipStream_t stream0;
	hipStream_t stream1;
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);
	unsigned int *a, *b, *c;
	unsigned int *da, *db, *dc;

	// paged-locked allocation
	hipHostAlloc((void**)&a, NO_LOOPS * MEMSIZE, hipHostMallocDefault);
	hipHostAlloc((void**)&b, NO_LOOPS * MEMSIZE, hipHostMallocDefault);
	hipHostAlloc((void**)&c, NO_LOOPS * MEMSIZE, hipHostMallocDefault);

	fillData(a, NO_LOOPS * N);
	fillData(b, NO_LOOPS * N);

	// Data chunks on GPU
	hipMalloc((void**)&da, MEMSIZE);
	hipMalloc((void**)&db, MEMSIZE);
	hipMalloc((void**)&dc, MEMSIZE);

	//T ODO: create stream

	unsigned int dataOffset0 = 0;
	unsigned int dataOffset1 = N;

	hipEvent_t startEvent, stopEvent;
	float elapsedTime;
	hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);
	hipEventRecord(startEvent, 0);

	for (int i = 0; i < NO_LOOPS; i += 2)
	{
		//TODO:  copy a->da, b->db
		//TODO:  run the kernel in the stream
		//TODO:  copy dc->c
		hipMemcpyAsync(da, &a[dataOffset0], MEMSIZE, hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(da, &a[dataOffset1], MEMSIZE, hipMemcpyHostToDevice, stream1);

		hipMemcpyAsync(db, &b[dataOffset0], MEMSIZE, hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(db, &b[dataOffset1], MEMSIZE, hipMemcpyHostToDevice, stream1);

		kernel << <GRID_SIZE, THREAD_PER_BLOCK, 0, stream0 >> > (da, db, N, dc);
		kernel << <GRID_SIZE, THREAD_PER_BLOCK, 0, stream1 >> > (da, db, N, dc);
		
		hipMemcpyAsync(&c[dataOffset0], dc, MEMSIZE, hipMemcpyDeviceToHost, stream0);
		hipMemcpyAsync(&c[dataOffset1], dc, MEMSIZE, hipMemcpyDeviceToHost, stream1);

		dataOffset0 += (2 * N);
		dataOffset1 += (2 * N);
	}

	//TODO: Synchonize stream

	hipStreamSynchronize(stream0);
	hipStreamSynchronize(stream1);

	//TODO: Destroy stream

	hipStreamDestroy(stream0);
	hipStreamDestroy(stream1);

	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);
	hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
	printf("\nTest 3 time: %f ms\n", elapsedTime);

	printData(c, 100);

	hipFree(da);
	hipFree(db);
	hipFree(dc);

	hipHostFree(a);
	hipHostFree(b);
	hipHostFree(c);
}


void exercise10()
{

	test1();
	test2();
	test3();

}
